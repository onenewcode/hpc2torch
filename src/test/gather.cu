#include "hip/hip_runtime.h"
#include <cuda_fp16.hpp>

template <typename T, int BLOCK_DIM_x, int numPerThreadx>
__global__ void kernel(T const *input, T *output,
                                   int othersize, int dimsize, int stride)
{

}
// TODO 目前进支持二维，且长度不超过1024
template <typename T>
void gatherLaunch(int const rank,int const shapesize,int const axis,int const *shape,void const *input,int const *index, void *output)
{
    
}

extern "C" void gather_f32(int const rank,int const shapesize,int const axis,int const *shape,void const *input,int const *index, void *output)
{
    gatherLaunch<float>(rank,shapesize,axis,shape,static_cast<float const*>(input),index, static_cast<float*>(output));
}
extern "C" void gather_f16(int const rank,int const shapesize,int const axis,int const *shape,void const *input,int const *index, void *output)
{
    gatherLaunch<half>(rank,shapesize,axis,shape,static_cast<half const*>(input),index, static_cast<half*>(output));
}